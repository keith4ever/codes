
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__global__ void avgpool2d_gelu_dequant(
                        int kernel_size, int stride, 
                        float min_value, float scale, 
                        int ch, int in_height, int in_width, 
                        int height, int width,
                        unsigned char *input, float *output)
{
    int idxC = threadIdx.x + blockIdx.x * blockDim.x; 
    int idxY = threadIdx.y + blockIdx.y * blockDim.y; 
    int idxX = threadIdx.z + blockIdx.z * blockDim.z; 
    if( idxC >= ch || idxY >= height || idxX >= width)
        return; 
    
    int sum = 0;
    int i = idxY * stride;
    int j = idxX * stride;
    // AvgPool2d op
    for(unsigned char m = 0; m < kernel_size; m++)
        for(unsigned char n = 0; n < kernel_size; n++){
            if((i + m) >= in_height || (j + n) >= in_width)
                continue; 
            sum += input[idxC * in_height * in_width + (i + m)*in_width + j + n];
        }
    // dequantize from int8 to float, along with division by kernel_size ^ 2
    float sum_deq = float((sum * scale) / (kernel_size * kernel_size)) + min_value;
    // GELU op
    output[idxC * height * width + idxY * width + idxX] =
        0.5 * sum_deq * (1.0 + erff(sum_deq / sqrt(2.0)));
}

